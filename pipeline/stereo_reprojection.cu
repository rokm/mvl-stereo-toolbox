#include "hip/hip_runtime.h"
/*
 * Stereo Pipeline: stereo reprojection
 * Copyright (C) 2013 Rok Mandeljc
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 * 
 */

#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/vec_traits.hpp>

__constant__ float cq[16];
__constant__ ushort off_x, off_y;

__global__ void reproject_kernel (const cv::gpu::PtrStepSz<unsigned char> disparity, cv::gpu::PtrStepSz<float3> points)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= disparity.rows || x >= disparity.cols) {
        return;
    }

    const float qx = (x + off_x) * cq[ 0] + (y + off_y) * cq[ 1] + cq[ 3];
    const float qy = (x + off_x) * cq[ 4] + (y + off_y) * cq[ 5] + cq[ 7];
    const float qz = (x + off_x) * cq[ 8] + (y + off_y) * cq[ 9] + cq[11];
    const float qw = (x + off_x) * cq[12] + (y + off_y) * cq[13] + cq[15];

    const unsigned char d = disparity(y, x);

    const float iW = 1.f / (qw + cq[14] * d);

    float3 v = cv::gpu::device::VecTraits<float3>::all(1.0f);
    v.x = (qx + cq[2] * d) * iW;
    v.y = (qy + cq[6] * d) * iW;
    v.z = (qz + cq[10] * d) * iW;

    points(y, x) = v;
}


void reprojectDisparityImageGpu (const cv::gpu::PtrStepSz<unsigned char> disparity, cv::gpu::PtrStepSz<float3> points, const float *q, unsigned short offsetX, unsigned short offsetY)
{
    dim3 block(32, 8);
    dim3 grid(cv::gpu::divUp(disparity.cols, block.x), cv::gpu::divUp(disparity.rows, block.y));

    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(off_x), &offsetX, sizeof(offsetX)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(off_y), &offsetY, sizeof(offsetY)));

    reproject_kernel<<<grid, block, 0>>>(disparity, points);
    cudaSafeCall(hipGetLastError());
}
